#include <cudpp.h>
#include <stdio.h>
#include "gpusort.h"

#define check_cuda_error() {\
	if (hipError_t e = hipGetLastError()) { \
		fprintf(stderr, "%s:%i: %s\n", __FILE__, __LINE__, \
				hipGetErrorString(e)); \
		exit(-1); \
	} }

void runSortingKernel(float *keys, int *values, int n) {
	float *d_keys = 0;
	int *d_values = 0;
	CUDPPConfiguration config;
	size_t keys_bytes = n * sizeof(*d_keys),
		   values_bytes = n * sizeof(*d_values);
	config.algorithm = CUDPP_SORT_RADIX;
	config.options = CUDPP_OPTION_KEY_VALUE_PAIRS;
	config.datatype = CUDPP_FLOAT;
	CUDPPHandle planhandle = 0;
	CUDPPResult result = cudppPlan(&planhandle, config, n, 1, 0);
	if (CUDPP_SUCCESS != result) {
		fprintf(stderr, "Error creating CUDPPPlan\n");
		exit(-1);
	}
	hipMalloc((void**) &d_keys, keys_bytes);
	check_cuda_error();
	hipMalloc((void**) &d_values, values_bytes);
	check_cuda_error();
	hipMemcpy(d_keys, keys, keys_bytes, hipMemcpyHostToDevice);
	check_cuda_error();
	hipMemcpy(d_values, values, values_bytes, hipMemcpyHostToDevice);
	check_cuda_error();
	cudppSort(planhandle, d_keys, d_values, sizeof(*d_keys) * 8, n);
	check_cuda_error();
	cudppDestroyPlan(planhandle);
	check_cuda_error();
	hipMemcpy(values, d_values, values_bytes, hipMemcpyDeviceToHost);
	check_cuda_error();
	hipDeviceSynchronize();
	check_cuda_error();
	hipFree(d_keys);
	hipFree(d_values);
}
